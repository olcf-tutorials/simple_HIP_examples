/* =========================================================================================
This program fills an NxN matrix (where N is even), A, with alternating sin(index) and 
cos(index) down its diagonal, performs the matrix multiply A*A on the GPU, then checks if 
the sum of the diagonal of the resulting matrix equals N/2 since 

    sin(index)*sin(index) + cos(index)*cos(index) = 1

Written by Tom Papatheodore
========================================================================================= */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <hipblas.h>

// Macro for checking errors in CUDA API calls
#define cudaErrorCheck(call)                                                              \
do{                                                                                       \
    hipError_t cuErr = call;                                                             \
    if(hipSuccess != cuErr){                                                             \
      printf("CUDA Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(cuErr));\
      exit(0);                                                                            \
    }                                                                                     \
}while(0)

// Size of matrices
#define N 512


int main(int argc, char *argv[])
{

	// Set device to GPU 0
	cudaErrorCheck( hipSetDevice(0) );


	/* Allocate memory for A, B on CPU -------------------------------------------------*/

	double *A = (double*)malloc(N*N*sizeof(double));
	double *B = (double*)malloc(N*N*sizeof(double));


	/* Set Values for A, B on CPU ------------------------------------------------------*/

	for(int i=0; i<N; i++){
		for(int j=0; j<N; j++){

            int index = i*N + j;

            if(i == j){
                if(i % 2 == 0){
                    A[index] = sin((double)index);
                }
                else{
                    A[index] = cos((double)index);
                }
            }

            B[index] = 0.0;
        }
	}


	/* Allocate memory for d_A, d_B on GPU ---------------------------------------------*/

	double *d_A, *d_B;
	cudaErrorCheck( hipMalloc(&d_A, N*N*sizeof(double)) );
	cudaErrorCheck( hipMalloc(&d_B, N*N*sizeof(double)) );


	/* Copy host arrays (A,B) to device arrays (d_A,d_B) -------------------------------*/

	cudaErrorCheck( hipMemcpy(d_A, A, N*N*sizeof(double), hipMemcpyHostToDevice) );
	cudaErrorCheck( hipMemcpy(d_B, B, N*N*sizeof(double), hipMemcpyHostToDevice) );


	/* Perform Matrix Multiply on GPU --------------------------------------------------*/

    const double alpha = 1.0;
    const double beta = 0.0;

    hipblasHandle_t handle;
    hipblasCreate(&handle);

	hipblasStatus_t status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_A, N, &beta, d_B, N);

	if (status != HIPBLAS_STATUS_SUCCESS){
		printf("hipblasDgemm failed with code %d\n", status);
		return EXIT_FAILURE;
	}


	/* Copy values of d_B (computed on GPU) into host array B --------------------------*/
	cudaErrorCheck( hipMemcpy(B, d_B, N*N*sizeof(double), hipMemcpyDeviceToHost) );

    /* Check if result is "roughly" N/2 ------------------------------------------------*/

    double sum       = 0.0;

    for(int i=0; i<N; i++){
        for(int j=0; j<N; j++){

            if(i == j){ sum += B[i*N + j]; }
        }
    }

    double difference = sum - N/2;
    if(difference > 0){
        sum = floor(sum); 
    }
    else if(difference < 0){
        sum = ceil(sum);
    }

    if(sum != N/2){
        printf("sum = %f instead of %f\n", sum, (double)(N/2));
        exit(-1);
    }

	/* Clean up and output --------------------------------------------------------------*/

	hipblasDestroy(handle);

    cudaErrorCheck( hipFree(d_A) );
    cudaErrorCheck( hipFree(d_B) );

    free(A);
    free(B);

    printf("__SUCCESS__\n");

    return 0;
}
